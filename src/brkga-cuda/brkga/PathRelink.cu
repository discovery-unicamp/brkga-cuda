#include "hip/hip_runtime.h"
#include "../BasicTypes.hpp"
#include "../Brkga.hpp"
#include "../Chromosome.hpp"
#include "../Decoder.hpp"
#include "../Logger.hpp"
#include "../utils/GpuUtils.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <numeric>
#include <random>
#include <stdexcept>
#include <string>
#include <utility>
#include <vector>

// FIXME this is an experimental feature

namespace box {
template <class T>
__global__ void copyChromosome(T* dst,
                               const uint index,
                               const T* src,
                               const uint chromosomeLength,
                               const uint* fitnessIdx) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= chromosomeLength) return;

  const auto sortedIndex = fitnessIdx[index];
  dst[k] = src[sortedIndex * chromosomeLength + k];
}

__global__ void copyFitness(box::Fitness* fitness,
                            uint index,
                            box::Fitness* dFitness,
                            uint* dFitnessIdx) {
  const auto sortedIndex = dFitnessIdx[index];
  *fitness = dFitness[sortedIndex];
}

template <class T>
__global__ void copyToDevice(T* dst,
                             const uint index,
                             const T* src,
                             const uint chromosomeLength,
                             const uint* fitnessIdx) {
  const auto k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= chromosomeLength) return;

  const auto sortedIndex = fitnessIdx[index];
  dst[sortedIndex * chromosomeLength + k] = src[k];
}

template <class T>
__host__ __device__ void setupBlock(uint j,
                                    Chromosome<T>* wrapper,
                                    T* chromosomes,
                                    const uint* blocks,
                                    uint blockSize,
                                    uint chromosomeLength,
                                    uint id) {
  const auto b = blocks[j];
  const auto l = b * blockSize;
  const auto r = l + blockSize;  // Overflow will never happen here
  assert(l < chromosomeLength);
  wrapper[j] = Chromosome<Gene>(chromosomes, chromosomeLength, /* base: */ id,
                                /* guide: */ (id ^ 1), l, r);
}

template <class T>
__global__ void buildBlocksKernel(Chromosome<T>* wrapper,
                                  T* chromosomes,
                                  const uint* blocks,
                                  uint blockSize,
                                  uint chromosomeLength,
                                  uint id) {
  const auto j = blockIdx.x * blockDim.x + threadIdx.x;
  setupBlock(j, wrapper, chromosomes, blocks, blockSize, chromosomeLength, id);
}

template <class T>
void buildBlocks(uint n,
                 Chromosome<T>* wrapper,
                 T* chromosomes,
                 const uint* blocks,
                 uint blockSize,
                 uint chromosomeLength,
                 uint id) {
  for (uint j = 0; j < n; ++j)
    setupBlock(j, wrapper, chromosomes, blocks, blockSize, chromosomeLength,
               id);
}

std::vector<Gene> Brkga::pathRelink(const uint base, const uint guide) {
  logger::debug("Running Path Relink with", base, "and", guide);

  auto dChromosomes = gpu::alloc<Gene>(nullptr, 2 * config.chromosomeLength());
  copyChromosome<<<gpu::blocks(config.chromosomeLength(), config.gpuThreads()),
                   config.gpuThreads()>>>(dChromosomes, base, dPopulation.get(),
                                          config.chromosomeLength(),
                                          dFitnessIdx.get());
  CUDA_CHECK_LAST();
  copyChromosome<<<gpu::blocks(config.chromosomeLength(), config.gpuThreads()),
                   config.gpuThreads()>>>(
      dChromosomes + config.chromosomeLength(), guide, dPopulation.get(),
      config.chromosomeLength(), dFitnessIdx.get());
  CUDA_CHECK_LAST();

  std::vector<Gene> chromosomes(2 * config.chromosomeLength());
  gpu::copy2h(nullptr, chromosomes.data(), dChromosomes,
              2 * config.chromosomeLength());
  gpu::sync();

  std::vector<Gene> bestGenes(chromosomes.begin(),
                              chromosomes.begin() + config.chromosomeLength());

  auto* dBestFitness = gpu::alloc<box::Fitness>(nullptr, 1);
  copyFitness<<<1, 1>>>(dBestFitness, base, dFitness.get(), dFitnessIdx.get());
  CUDA_CHECK_LAST();
  box::Fitness bestFitness = -1e30f;
  gpu::copy2h(nullptr, &bestFitness, dBestFitness, 1);
  gpu::free(nullptr, dBestFitness);
  logger::debug("Starting PR with:", bestFitness);

  const auto numberOfSegments =
      (config.chromosomeLength() + config.pathRelinkBlockSize() - 1)
      / config.pathRelinkBlockSize();
  logger::debug("Number of blocks to process:", numberOfSegments);
  std::vector<uint> blocks(numberOfSegments);
  std::iota(blocks.begin(), blocks.end(), 0);

  fitness.resize(numberOfSegments);

  uint* dBlocks = nullptr;
  box::Fitness* dFitnessPtr = nullptr;
  if (!config.decodeType().onCpu()) {
    dBlocks = gpu::alloc<uint>(nullptr, numberOfSegments);
    dFitnessPtr = gpu::alloc<box::Fitness>(nullptr, numberOfSegments);
  }

  uint id = 0;
  for (uint i = numberOfSegments; i > 0; --i) {
    if (config.decodeType().onCpu()) {
      buildBlocks(i, populationWrapper, chromosomes.data(), blocks.data(),
                  config.pathRelinkBlockSize(), config.chromosomeLength(), id);
      config.decoder()->decode(i, populationWrapper, fitness.data());
    } else {
      gpu::copy2d(streams[0], dChromosomes, chromosomes.data(),
                  chromosomes.size());
      gpu::copy2d(streams[0], dBlocks, blocks.data(), i);
      buildBlocksKernel<<<1, (unsigned)i, 0, streams[0]>>>(
          populationWrapper, dChromosomes, dBlocks,
          config.pathRelinkBlockSize(), config.chromosomeLength(), id);
      config.decoder()->decode(streams[0], i, populationWrapper, dFitnessPtr);
      gpu::copy2h(streams[0], fitness.data(), dFitnessPtr, i);
      gpu::sync(streams[0]);
    }

    uint bestIdx = 0;
    for (uint j = 1; j < i; ++j) {
      if (fitness[j] < fitness[bestIdx]) bestIdx = j;
    }
    logger::debug("PR moved to:", fitness[bestIdx],
                  format(Separator(""), "(incumbent: ", bestFitness, ")"));

    const auto baseBegin = chromosomes.begin() + id * config.chromosomeLength();
    const auto guideBegin =
        chromosomes.begin() + (id ^ 1) * config.chromosomeLength();

    const auto changeOffset = blocks[bestIdx] * config.pathRelinkBlockSize();
    const auto bs = std::min(config.chromosomeLength() - changeOffset,
                             config.pathRelinkBlockSize());
    auto itFrom = guideBegin + changeOffset;
    auto itTo = baseBegin + changeOffset;
    std::copy(itFrom, itFrom + bs, itTo);

    if (fitness[bestIdx] < bestFitness) {
      bestFitness = fitness[bestIdx];
      std::copy(baseBegin, baseBegin + config.chromosomeLength(),
                bestGenes.begin());
    }

    std::swap(blocks[bestIdx], blocks[i - 1]);  // "Erase" the block used
    id ^= 1;  // "Swap" the base and the guide chromosome
  }

  logger::debug("Path Relink finished with:", bestFitness);

  gpu::free(nullptr, dChromosomes);
  gpu::free(nullptr, dBlocks);
  return bestGenes;
}

void Brkga::runPathRelink(const std::vector<PathRelinkPair>& pairList) {
  // TODO move the chromosomes based on the order of their fitness (for GPU)
  //   is it really necessary to move now?
  // TODO ensure population wrapper has enough capacity
  // FIXME add support to permutation
  // TODO can we implement this for the permutation without sorting every time?
  logger::debug("Run Path Relink between", pairList.size(), "pairs");
  if (config.pathRelinkBlockSize() == 0)
    throw InvalidArgument("Block size wasn't defined", __FUNCTION__);

  for (const auto& pair : pairList) {
    InvalidArgument::max(
        Arg<uint>(pair.basePopulationId, "base population"),
        Arg<uint>(config.numberOfPopulations() - 1, "#populations - 1"),
        __FUNCTION__);
    InvalidArgument::max(
        Arg<uint>(pair.guidePopulationId, "guide population"),
        Arg<uint>(config.numberOfPopulations() - 1, "#populations - 1"),
        __FUNCTION__);
    InvalidArgument::max(
        Arg<uint>(pair.baseChromosomeId, "base chromosome"),
        Arg<uint>(config.populationSize() - 1, "|population| - 1"),
        __FUNCTION__);
    InvalidArgument::max(
        Arg<uint>(pair.guideChromosomeId, "guide chromosome"),
        Arg<uint>(config.populationSize() - 1, "|population| - 1"),
        __FUNCTION__);
  }

  std::vector<uint> insertedCount(config.numberOfPopulations(), 0);
  auto dChromosomes = gpu::alloc<Gene>(nullptr, config.chromosomeLength());

  for (const auto& pair : pairList) {
    const auto base =
        pair.basePopulationId * config.populationSize() + pair.baseChromosomeId;
    const auto guide = pair.guidePopulationId * config.populationSize()
                       + pair.guideChromosomeId;

    const auto bestGenes = pathRelink(base, guide);

    // TODO use hamming distance/kendall tau to check if it should be included?
    //   maybe give the user a method to filter "duplicated" chromosomes with
    //   those methods

    ++insertedCount[pair.basePopulationId];
    assert(insertedCount[pair.basePopulationId]
           < config.populationSize() - config.numberOfElites());
    const auto replacedChromosomeIndex =
        config.populationSize() - insertedCount[pair.basePopulationId];

    logger::debug("Copying the chromosome found back to the device");
    gpu::copy2d(nullptr, dChromosomes, bestGenes.data(),
                config.chromosomeLength());
    copyToDevice<<<gpu::blocks(config.chromosomeLength(), config.gpuThreads()),
                   config.gpuThreads()>>>(
        dPopulation.row(pair.basePopulationId), replacedChromosomeIndex,
        dChromosomes, config.chromosomeLength(),
        dFitnessIdx.row(pair.basePopulationId));
    CUDA_CHECK_LAST();
  }

  gpu::free(nullptr, dChromosomes);

  // FIXME should decode only the new chromosomes, not the population
  updateFitness();
  logger::debug("The Path Relink has finished");
}
}  // namespace box
