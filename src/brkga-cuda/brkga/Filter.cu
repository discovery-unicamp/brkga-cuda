#include "hip/hip_runtime.h"
#include "../BasicTypes.hpp"
#include "../Brkga.hpp"
#include "../Chromosome.hpp"
#include "../Comparator.hpp"
#include "../Logger.hpp"
#include "../utils/GpuUtils.hpp"

#include <set>
#include <vector>

namespace box {
__global__ void copySorted(Gene* sortedPopulation,
                           const unsigned* fitnessIdx,
                           const Gene* population,
                           unsigned numberOfPopulations,
                           unsigned populationSize,
                           unsigned chromosomeLength) {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= numberOfPopulations * populationSize) return;

  const auto p = tid / populationSize;
  const auto c = tid % populationSize;
  const auto* from = population
                     + (p * populationSize + fitnessIdx[p * populationSize + c])
                           * chromosomeLength;
  auto* to = sortedPopulation + tid * chromosomeLength;

  for (unsigned i = 0; i < chromosomeLength; ++i) to[i] = from[i];
}

void Brkga::printStatus() {
  logger::debug("Copy chromosomes sorted");
  copySorted<<<gpu::blocks(
                   config.numberOfPopulations() * config.populationSize(),
                   config.gpuThreads()),
               config.gpuThreads()>>>(
      dPopulationTemp.get(), dFitnessIdx.get(), dPopulation.get(),
      config.numberOfPopulations(), config.populationSize(),
      config.chromosomeLength());
  CUDA_CHECK_LAST();
  gpu::sync();

  logger::debug("Copy data to host");
  assert(config.decodeType().chromosome());
  population.resize(config.numberOfPopulations() * config.populationSize()
                    * config.chromosomeLength());
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    gpu::copy2h(streams[p],
                population.data()
                    + p * config.populationSize() * config.chromosomeLength(),
                dPopulation.row(p),
                config.populationSize() * config.chromosomeLength());
  }
  syncStreams();

  logger::debug("Print info");
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    unsigned k = 0;
    for (unsigned i = 0; i < config.numberOfElites(); i += k) {
      for (k = 1; i + k < config.populationSize(); ++k) {
        const auto* ci = population.data()
                         + (p * config.populationSize() + i + k - 1)
                               * config.chromosomeLength();
        const auto* ck =
            population.data()
            + (p * config.populationSize() + i + k) * config.chromosomeLength();

        const float eps = 1e-6f;
        bool eq = true;
        for (unsigned j = 0; j < config.chromosomeLength(); ++j) {
          if (std::abs(ci[j] - ck[j]) >= eps) {
            eq = false;
            break;
          }
        }

        if (!eq) break;
      }

      if (k > 2) {
        logger::warning("Found", k, "equal chromosomes on population", p);
      }
    }
  }
}

void Brkga::removeSimilarElites(const ComparatorBase& comparator) {
  logger::debug("Removing duplicated chromosomes");

  // FIXME this block was duplicated
  population.resize(config.numberOfPopulations() * config.populationSize()
                    * config.chromosomeLength());
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    gpu::copy2h(streams[p],
                population.data()
                    + p * config.populationSize() * config.chromosomeLength(),
                dPopulation.row(p),
                config.populationSize() * config.chromosomeLength());
  }

  // TODO should i update the fitness too?
  // fitness.resize(config.numberOfPopulations() * config.populationSize());
  // for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
  //   gpu::copy2h(streams[p], fitness.data() + p * config.populationSize(),
  //                dFitness.row(p), config.populationSize());
  // }

  std::vector<unsigned> fitnessIdx(
      config.numberOfPopulations() * config.populationSize(), -1u);
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    gpu::copy2h(streams[p], fitnessIdx.data() + p * config.populationSize(),
                dFitnessIdx.row(p), config.populationSize());
  }

  syncStreams();

  unsigned duplicatedCount = 0;
  // TODO replace by the worst fitness * factor
  // const float badFitness = 1e18;

  std::vector<Chromosome<Gene>> elites(config.numberOfElites());
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    logger::debug("Pruning population", p);
    const auto offset = p * config.populationSize();

    for (unsigned i = 0; i < config.numberOfElites(); ++i) {
      elites[i] = Chromosome<Gene>(
          population.data() + offset * config.chromosomeLength(),
          config.chromosomeLength(), fitnessIdx[offset + i]);
    }

    unsigned k = 0;
    std::vector<unsigned> removedIdx;
    std::vector<bool> remove(config.numberOfElites(), false);
    for (unsigned i = 0; i < config.numberOfElites(); ++i) {
      if (remove[i]) {
        // fitness[fitnessIdx[offset + i]] = badFitness;
        removedIdx.push_back(fitnessIdx[offset + i]);
        continue;
      }

      fitnessIdx[offset + k] = fitnessIdx[offset + i];
      ++k;
      for (unsigned j = i + 1; j < config.numberOfElites(); ++j)
        remove[j] = remove[j] || comparator(elites[i], elites[j]);
    }
    if (removedIdx.empty()) continue;
    duplicatedCount += (unsigned)removedIdx.size();

    // TODO is this enough?
    for (unsigned i = config.numberOfElites(); i < config.populationSize();
         ++i) {
      fitnessIdx[offset + k] = fitnessIdx[offset + i];
      ++k;
    }
    for (unsigned idx : removedIdx) {
      fitnessIdx[offset + k] = idx;
      ++k;
    }
    assert(k == config.populationSize());
    assert((unsigned)std::set<unsigned>(
               fitnessIdx.begin() + offset,
               fitnessIdx.begin() + offset + config.populationSize())
               .size()
           == config.populationSize());
  }

  logger::debug("Copying data to device");
  for (unsigned p = 0; p < config.numberOfPopulations(); ++p) {
    gpu::copy2d(streams[p], dFitnessIdx.row(p),
                fitnessIdx.data() + p * config.populationSize(),
                config.populationSize());
  }

  logger::debug("Removed", duplicatedCount, "duplicated chromosomes");
}
}  // namespace box
